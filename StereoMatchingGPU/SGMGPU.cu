#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<string>
#include<iostream>
#include<fstream>
#include<vector>
#include <hip/hip_runtime.h>
#include<hip/hip_vector_types.h>
#include<>
#include"FreeImage.h"

#define PenaltySmall	10
#define PenaltyLarge	50
#define PATHTOTAL		8
#define PAD16(x)		((x+15)&(~15))
struct f4
{
	f4() { x = y = z = w = 0.f; }
	f4(float _x, float _y, float _z, float _w) :
		x(_x), y(_y), z(_z), w(_w) {}
	float x, y, z, w;
};

int	clampIndex(int x, int Min, int Max)
{
	if (x < Min) return Min;
	else if (x > Max) return Max;
	else return x;
}


f4* loadImage(const std::string& filename,
	int& width, int& height)
{
	FreeImage_Initialise(true);

	FIBITMAP* bmpConverted = nullptr;
	FREE_IMAGE_FORMAT fif = FIF_UNKNOWN;
	//��ȡ�ļ�����
	fif = FreeImage_GetFileType(filename.c_str());
	if (fif == FIF_UNKNOWN)
		fif = FreeImage_GetFIFFromFilename(filename.c_str());

	if (fif == FIF_UNKNOWN)
	{
		std::cout << "�����ʽδ֪���޷�����" << std::endl;
		return false;
	}
	if (!FreeImage_FIFSupportsReading(fif))
	{
		std::cout << "�����ʽ����֧�֣��޷�����" << std::endl;
		return false;
	}

	//�������� ���
	FIBITMAP* dib = FreeImage_Load(fif, filename.c_str());
	if (!dib)
	{
		std::cout << "�������ʧ�ܣ�" << std::endl;
		return false;
	}

	//��jmxR�У����Ͻ�Ϊ����(0,0)
	//������Ҫ��������

	FreeImage_FlipVertical(dib);

	width = FreeImage_GetWidth(dib);
	height = FreeImage_GetHeight(dib);

	//�����������ݽṹ
	auto data = new f4[width*height];

	RGBQUAD rgb;
	for (int y = 0; y < height; ++y)
	{
		for (int x = 0; x < width; ++x)
		{
			FreeImage_GetPixelColor(dib, x, y, &rgb);

			//rgb�У�ÿ������ΪBYTE����(0~255)
			//����ת��Ϊ[0,1]��f32�ͣ���������Spectrum��
			float r, g, b;
			r = float(rgb.rgbRed);
			g = float(rgb.rgbGreen);
			b = float(rgb.rgbBlue);
			data[y*width + x] = f4(r, g, b, 0.f);
		}
	}

	//�ͷž��
	FreeImage_Unload(dib);

	return data;
}
//8
int*	devPathDir;
int*	devWidth;
int*	devHeight;
int*	devDint;
//width*height
float*	devLeftGray;
//width*height
float*  devRightGray;


//width*height*dint
float*	devImageCost;
float*	devImageInterCost;
//width*height*dint*PathTotal
float*	devPathCost;
//width*height*dint
float*  devImageS;
//width*height*PathTotal
float*	devPathMin;
//width*height
int*    devDisparity;

int		hostWidth;
int		hostHeight;
int		hostDint;
int*	hostDisparity;
f4*		hostLeft;
f4*		hostRight;


__device__ int	cudaClampIndex(int x, int Min, int Max)
{
	if (x < Min) return Min;
	else if (x > Max) return Max;
	else return x;
}

__device__ int	cudaMinI(int a, int b)
{
	return a < b ? a : b;
}

__device__ float cudaMinF(float a, float b)
{
	return a < b ? a : b;
}


__device__ int cudaMaxI(int a, int b)
{
	return a > b ? a : b;
}
__device__ float cudaMaxF(float a, float b)
{
	return a > b ? a : b;
}


__device__ float cudaAbs(float a)
{
	return a > 0.f ? a : -a;
}

__device__ float interHalf(int x, int y, float* image, float iq,
	int width,int height,int dint)
{
	int xplusone = cudaMinI(width - 1, x + 1);
	int xsubone = cudaMaxI(0, x - 1);

	int index0 = y*width + x;
	int index1 = y*width + xplusone;
	int index2 = y*width + xsubone;

	float dpq = cudaMinI(cudaAbs(image[index0] - iq), cudaAbs((image[index0] + image[index1]) / 2 - iq));
	dpq = cudaMinI(dpq, cudaAbs((image[index0] + image[index2]) / 2 - iq));

	return dpq;
}

__global__ void calCost(float* imageCost,float* left,float*right,float* imageS,float* pathCost,float* pathMin,
	int* pwidth,int* pheight,int* pdint)
{
	int x = threadIdx.x + blockIdx.x*blockDim.x;
	int y = threadIdx.y + blockIdx.y*blockDim.y;
	
	int width = *pwidth;
	int height = *pheight;
	int dint = *pdint;

	if (x >= width || y >= height ) return;
	


	//pathRowCost �����·���ϵ�L(p,d)   width*height**dint*pathtotal
	//pathRowMinCost �����·���ϵ�Lmin(p,d)  width*height*pathtotal
	auto pathCostStride = width*height*dint;
	auto pathRowCost = pathCost;
	auto pathMinStride = width*height;
	auto pathRowMinCost = pathMin;
	float minCost = 1 << 21;

	//��ʼ��pathMin
	auto imageIndex = y*width + x;
	for (int p = 0;p < PATHTOTAL;++p)
	{
		pathRowMinCost = pathMin + p*pathMinStride;
		pathRowMinCost[imageIndex] = 1 << 21;
	}

	
	for (int d = 0;d < dint;++d)
	{
		//����cost
		auto costIndex = y*width*dint + x*dint + d;
		auto cost = 0.f;
		if (d >= x)
		{
			cost = (float)(1 << 20);
		}
		else
		{
			cost = cudaMinF(interHalf(x, y, left, right[imageIndex - d], width, height, dint),
				interHalf(x - d, y, right, left[imageIndex], width, height, dint));
		}
		imageCost[costIndex] = cost;

		minCost = cudaMinF(minCost, cost);

		//��ʼ��imageS
		imageS[costIndex] = 0;


		//��ʼ��pathCost
		for (int p = 0;p < PATHTOTAL;++p)
		{
			
			pathRowCost = pathCost + p*pathCostStride;
			pathRowCost[costIndex] = 1 << 20;

			
		}

		//pathCost�߽��ж�
		if (x == 0 || y == 0 || x == width - 1 || y == height - 1)
		{
			pathRowCost = pathCost + 0 * pathCostStride;
			pathRowCost[costIndex] = cost;

			pathRowCost = pathCost + 1 * pathCostStride;
			pathRowCost[costIndex] = cost;

			pathRowCost = pathCost + 2 * pathCostStride;
			pathRowCost[costIndex] = cost;

			pathRowCost = pathCost + 3 * pathCostStride;
			pathRowCost[costIndex] = cost;

			pathRowCost = pathCost + 4 * pathCostStride;
			pathRowCost[costIndex] = cost;

			pathRowCost = pathCost + 5 * pathCostStride;
			pathRowCost[costIndex] = cost;

			pathRowCost = pathCost + 6 * pathCostStride;
			pathRowCost[costIndex] = cost;

			pathRowCost = pathCost + 7 * pathCostStride;
			pathRowCost[costIndex] = cost;
			

			imageS[costIndex] += cost * 8;
		}

		
	}//end for d

	//pathMin�߽��ж�
	if (x == 0 || y == 0 || x == width - 1 || y == height - 1)
	{
		pathRowMinCost = pathMin + 0 * pathMinStride;
		pathRowMinCost[imageIndex] = minCost;

		pathRowMinCost = pathMin + 1 * pathMinStride;
		pathRowMinCost[imageIndex] = minCost;

		pathRowMinCost = pathMin + 2 * pathMinStride;
		pathRowMinCost[imageIndex] = minCost;

		pathRowMinCost = pathMin + 3 * pathMinStride;
		pathRowMinCost[imageIndex] = minCost;

		pathRowMinCost = pathMin + 4 * pathMinStride;
		pathRowMinCost[imageIndex] = minCost;

		pathRowMinCost = pathMin + 5 * pathMinStride;
		pathRowMinCost[imageIndex] = minCost;

		pathRowMinCost = pathMin + 6 * pathMinStride;
		pathRowMinCost[imageIndex] = minCost;

		pathRowMinCost = pathMin + 7 * pathMinStride;
		pathRowMinCost[imageIndex] = minCost;
	}

	

}

__global__ void subPixelInter(float* imageCost, float* imageInterCost,
	int* pwidth, int* pheight, int* pdint)
{
	int x = threadIdx.x + blockIdx.x*blockDim.x;
	int y = threadIdx.y + blockIdx.y*blockDim.y;
	int d = threadIdx.z + blockIdx.z*blockDim.z;
	

	int width = *pwidth;
	int height = *pheight;
	int dint = *pdint;
	if (x >= width || y >= height || d >= dint) return;

	auto costIndex = y*width*dint + x*dint + d;

	auto total = 0.f;

	int weight = 0;
	for (int oy = -5;oy < 5;++oy)
	{
		int ny = cudaClampIndex(y + oy, 0, height - 1);
		for (int ox = -5;ox < 5;++ox)
		{
			int nx = cudaClampIndex(x + ox, 0, width - 1);

			int index = ny*width*dint + nx*dint + d;

			total += imageCost[index];
			weight++;
		}
	}

	imageInterCost[costIndex] = total / weight;
		
}

/*
__global__ void updatePath(float* imageCost, float* imageS, float* pathCost,float* pathMin,
	float* pwidth, float* pheight, float* pdint,int x,int* path)
{
	int y = threadIdx.x + blockIdx.x*blockDim.x;
	int p = threadIdx.y;
	int width = *pwidth;
	int height = *pheight;
	int dint = *pdint;
	if (x >= width - 1 || y >= height - 1 || x < 1 || y < 1) return;

	auto selectPathCost = pathCost + p*(width*height*dint);
	auto selectPathMin = pathMin+p*(width*height);


	auto xstep = path[p];
	auto ystep = path[p];
	auto adjx = x + xstep;
	auto adjy = y + ystep;
	auto adjMin = selectPathMin[y*width + x];


	for (int d = 0;d < dint;++d)
	{
		auto costIndex = y*width*dint + x*dint + d;
		auto imageIndex = y*width + x;


		auto dplusone = cudaMinI(d + 1, dint - 1);
		auto dminusone = cudaMaxI(d - 1, 0);

		auto min0 = cudaMinF(selectPathCost[adjy*width*dint + adjx*dint + d],
			selectPathCost[adjy*width*dint + adjx*dint + dplusone] + PenaltySmall);
		auto min1 = cudaMinF(min0, selectPathCost[adjy*width*dint + adjx*dint + dminusone] + PenaltySmall);
		auto min2 = cudaMinF(min1, adjMin + PenaltyLarge);

		auto cost = imageCost[costIndex] + min2 - adjMin;

		selectPathCost[costIndex] = cost;
		if (cost < selectPathMin[imageIndex])
		{
			selectPathMin[imageIndex] = cost;
		}

		imageS[costIndex] += cost;
	}
}


__global__ void updatePath2(float* imageCost, float* imageS, float* pathCost, float* pathMin,
	float* pwidth, float* pheight, float* pdint, int y, int pi, int* path)
{
	int x = threadIdx.x + blockIdx.x*blockDim.x;
	int p = threadIdx.y ;
	int width = *pwidth;
	int height = *pheight;
	int dint = *pdint;
	if (x >= width - 1 || y >= height - 1 || x < 1 || y < 1) return;

	auto selectPathCost = pathCost + p*(width*height*dint);
	auto selectPathMin = pathMin + p*(width*height);


	auto xstep = path[p];
	auto ystep = path[p];
	auto adjx = x + xstep;
	auto adjy = y + ystep;
	auto adjMin = selectPathMin[y*width + x];


	for (int d = 0;d < dint;++d)
	{
		auto costIndex = y*width*dint + x*dint + d;
		auto imageIndex = y*width + x;


		auto dplusone = cudaMinI(d + 1, dint - 1);
		auto dminusone = cudaMaxI(d - 1, 0);

		auto min0 = cudaMinF(selectPathCost[adjy*width*dint + adjx*dint + d],
			selectPathCost[adjy*width*dint + adjx*dint + dplusone] + PenaltySmall);
		auto min1 = cudaMinF(min0, selectPathCost[adjy*width*dint + adjx*dint + dminusone] + PenaltySmall);
		auto min2 = cudaMinF(min1, adjMin + PenaltyLarge);

		auto cost = imageCost[costIndex] + min2 - adjMin;

		selectPathCost[costIndex] = cost;
		if (cost < selectPathMin[imageIndex])
		{
			selectPathMin[imageIndex] = cost;
		}

		imageS[costIndex] += cost;
	}
}
*/

__global__ void forwardPass(float* imageCost, float* imageS, float* pathCost, float* pathMin,
	int* pwidth, int* pheight, int* pdint, int x, int* path)
{
	int y = threadIdx.y + blockIdx.y*blockDim.y;
	int width = *pwidth;
	int height = *pheight;
	int dint = *pdint;
	if (y == 0 || y == height - 1 || x >= width || y >= height) return;

	auto imageIndex = y*width + x;
	for (int d = 0;d < dint;++d)
	{
		auto costIndex = y*width*dint + x*dint + d;
		//pathRowCost �����·���ϵ�L(p,d)   width*height**dint*pathtotal
		//pathRowMinCost �����·���ϵ�Lmin(p,d)  width*height*pathtotal
		auto pathCostStride = width*height*dint;
		auto pathRowCost = pathCost;
		auto pathMinStride = width*height;
		auto pathRowMinCost = pathMin;

		//foward�׶� ��ǰ3��·��
		for (int p = 0;p < 3;++p)
		{
			pathRowCost = pathCost + p*pathCostStride;
			pathRowMinCost = pathMin + p*pathMinStride;


			//�����·���ϵ����ڵ�
			auto xstep = path[p * 2];
			auto ystep = path[p * 2 + 1];
			auto adjx = x + xstep;
			auto adjy = y + ystep;
			auto adjMin = pathRowMinCost[adjy*width + adjx];

			auto dplusone = cudaMinI(d + 1, dint - 1);
			auto dminusone = cudaMaxI(d - 1, 0);

			auto min0 = cudaMinF(pathRowCost[adjy*width*dint + adjx*dint + d],
				pathRowCost[adjy*width*dint + adjx*dint + dplusone] + PenaltySmall);
			auto min1 = cudaMinF(min0, pathRowCost[adjy*width*dint + adjx*dint + dminusone] + PenaltySmall);
			auto min2 = cudaMinF(min1, adjMin + PenaltyLarge);

			auto cost = imageCost[costIndex] + min2 - adjMin;

			pathRowCost[costIndex] = cost;
			if (cost < pathRowMinCost[imageIndex])
			{
				pathRowMinCost[imageIndex] = cost;
			}

			imageS[costIndex] += cost;

		}
	}
}


__global__ void backwardPass(float* imageCost, float* imageS, float* pathCost, float* pathMin,
	int* pwidth, int* pheight, int* pdint, int x, int* path)
{
	int y = threadIdx.y + blockIdx.y*blockDim.y;
	int width = *pwidth;
	int height = *pheight;
	int dint = *pdint;
	if (y == 0 || y == height - 1 || x >= width || y >= height) return;

	auto imageIndex = y*width + x;
	for (int d = 0;d < dint;++d)
	{
		auto costIndex = y*width*dint + x*dint + d;

		//pathRowCost �����·���ϵ�L(p,d)   width*height**dint*pathtotal
		//pathRowMinCost �����·���ϵ�Lmin(p,d)  width*height*pathtotal

		auto pathCostStride = width*height*dint;
		auto pathRowCost = pathCost;
		auto pathMinStride = width*height;
		auto pathRowMinCost = pathMin;

		//backword�׶� ,4~7��ŵ�·��
		for (int p = 4;p < 7;++p)
		{
			pathRowCost = pathCost + p*pathCostStride;
			pathRowMinCost = pathMin + p*pathMinStride;


			//�����·���ϵ����ڵ�
			auto xstep = path[p * 2];
			auto ystep = path[p * 2 + 1];
			auto adjx = x + xstep;
			auto adjy = y + ystep;
			auto adjMin = pathRowMinCost[adjy*width + adjx];

			auto dplusone = cudaMinI(d + 1, dint - 1);
			auto dminusone = cudaMaxI(d - 1, 0);

			auto min0 = cudaMinF(pathRowCost[adjy*width*dint + adjx*dint + d],
				pathRowCost[adjy*width*dint + adjx*dint + dplusone] + PenaltySmall);
			auto min1 = cudaMinF(min0, pathRowCost[adjy*width*dint + adjx*dint + dminusone] + PenaltySmall);
			auto min2 = cudaMinF(min1, adjMin + PenaltyLarge);

			auto cost = imageCost[costIndex] + min2 - adjMin;

			pathRowCost[costIndex] = cost;
			if (cost < pathRowMinCost[imageIndex])
			{
				pathRowMinCost[imageIndex] = cost;
			}

			imageS[costIndex] += cost;

		}
	}
}


__global__ void calDisparity(float* imageS, int* disparity,
	int* pwidth,int *pheight,int* pdint)
{
	int x = threadIdx.x + blockIdx.x*blockDim.x;
	int y = threadIdx.y + blockIdx.y*blockDim.y;


	int width = *pwidth;
	int height = *pheight;
	int dint = *pdint;

	if (x >= width || y >= height) return;
	auto imageIndex = y*width + x;

	float minValue = float(1 << 23);
	int dindex = -1;
	for (int d = 1;d < dint;++d)
	{
		auto costIndex = y*width*dint + x*dint + d;
		auto s = imageS[costIndex];
		if (s < minValue)
		{
			minValue = s;
			dindex = d;
		}
	}
	disparity[imageIndex] = dindex;
}


void	covertGray(f4* left, f4* right,
	float** leftGray, float** rightGray)
{
	int wh = hostWidth*hostHeight;
	*leftGray = new float[wh];
	*rightGray = new float[wh];
	for (int i = 0;i < wh;++i)
	{
		auto c0 = left[i];
		(*leftGray)[i] = 0.212671f*c0.x + 0.715160f*c0.y + 0.072169f*c0.z;
		auto c1 = right[i];
		(*rightGray)[i] = 0.212671f*c1.x + 0.715160f*c1.y + 0.072169f*c1.z;
	}

}


void		createGPUBuffer()
{
	hipMalloc(&devPathDir, 8 * sizeof(int) * 2);
	hipMalloc(&devWidth, sizeof(int));
	hipMalloc(&devHeight, sizeof(int));
	hipMalloc(&devDint, sizeof(int));
	hipMalloc(&devLeftGray, hostWidth*hostHeight * sizeof(float));
	hipMalloc(&devRightGray, hostWidth*hostHeight * sizeof(float));
	hipMalloc(&devImageCost, hostWidth*hostHeight*hostDint*sizeof(float));
	hipMalloc(&devImageInterCost, hostWidth*hostHeight*hostDint * sizeof(float));
	hipMalloc(&devImageS, hostWidth*hostHeight*hostDint*sizeof(float));
	hipMalloc(&devPathCost, hostWidth*hostHeight*hostDint*PATHTOTAL*sizeof(float));
	hipMalloc(&devPathMin, hostWidth*hostHeight*PATHTOTAL*sizeof(float));
	hipMalloc(&devDisparity, hostWidth*hostHeight*sizeof(int));


	//CPU�� ��ĳЩBuffer���г�ʼ��
	std::vector<int> pathDir =
	{ -1,-1,
	-1,0,
	-1,+1,
	0, -1,
	+1,-1,
	+1,0,
	+1,+1,
	0,+1 };
	hipMemcpy(devPathDir, &pathDir[0], PATHTOTAL * 2 * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(devWidth, &hostWidth, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(devHeight, &hostHeight, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(devDint, &hostDint, sizeof(int), hipMemcpyHostToDevice);

	float* leftGray;
	float* rightGray;
	covertGray(hostLeft, hostRight, &leftGray, &rightGray);
	hipMemcpy(devLeftGray, leftGray, hostWidth*hostHeight * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(devRightGray, rightGray, hostWidth*hostHeight * sizeof(float), hipMemcpyHostToDevice);

	
	std::vector<float> hostMin(hostWidth*hostHeight, (float)(1 << 21));
	auto stride = hostWidth*hostHeight;
	for (int p = 0;p < PATHTOTAL;++p)
	{
		hipMemcpy(devPathMin + p*stride, &hostMin[0], stride * sizeof(float), hipMemcpyHostToDevice);
	}

	delete[] leftGray;
	delete[] rightGray;


	hostDisparity = new int[hostWidth*hostHeight];
	
}

void		freeGPUBuffer()
{
	hipFree(devPathDir);
	hipFree(devWidth);
	hipFree(devHeight);
	hipFree(devDint);
	hipFree(devLeftGray);
	hipFree(devRightGray);
	hipFree(devImageCost);
	hipFree(devImageInterCost);
	hipFree(devImageS);
	hipFree(devPathCost);
	hipFree(devPathMin);
	hipFree(devDisparity);
}






void  hostCalCost()
{
	//����Cost
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	auto width16 = PAD16(hostWidth);
	auto height16 = PAD16(hostHeight);

	dim3 blockNum(width16 / 16, height16 / 16);
	dim3 threadsPerBlock(16, 16);

	calCost<<<blockNum,threadsPerBlock>>>(devImageCost, devLeftGray, devRightGray, devImageS,
		devPathCost, devPathMin,
		devWidth, devHeight, devDint);


	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float deltaTime;
	hipEventElapsedTime(&deltaTime, start, stop);
	std::cout << "Cost���� ��ʱ: " << deltaTime <<"ms"<<std::endl;

}


void hostCalInterCost()
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);


	auto width16 = PAD16(hostWidth);
	auto height16 = PAD16(hostHeight);
	auto dint16 = PAD16(hostDint);

	dim3 blockNum(width16 / 16, height16 / 16, dint16 / 16);
	dim3 threadsPerBlock(16, 16, 16);

	subPixelInter << <blockNum, threadsPerBlock >> > (devImageCost, devImageInterCost,
		devWidth, devHeight, devDint);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float deltaTime;
	hipEventElapsedTime(&deltaTime, start, stop);
	std::cout << "InterCost���� ��ʱ: " << deltaTime << "ms" << std::endl;
}
void hostDP(float* devCost)
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	auto height16 = PAD16(hostHeight);
	dim3 blockNum(1, height16 / 16);
	dim3 threadsPerBlock(1, 16);
	
	for (int x = 1;x < hostWidth-1;++x)
	{
		
		forwardPass<<<blockNum,threadsPerBlock>>>(devCost, devImageS, devPathCost, devPathMin,
			devWidth, devHeight, devDint,
			x, devPathDir);

		backwardPass << <blockNum, threadsPerBlock >> > (devCost, devImageS, devPathCost, devPathMin,
			devWidth, devHeight, devDint,
			hostWidth - 1 - x, devPathDir);
	}

	

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float deltaTime;
	hipEventElapsedTime(&deltaTime, start, stop);
	std::cout << "DP���� ��ʱ: " << deltaTime << "ms" << std::endl;
}

void hostCalDisparity()
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	auto width16 = PAD16(hostWidth);
	auto height16 = PAD16(hostHeight);

	dim3 blockNum(width16 / 16, height16 / 16);
	dim3 threadsPerBlock(16, 16);

	calDisparity << < blockNum, threadsPerBlock >> > (devImageS, devDisparity,
		devWidth, devHeight, devDint);


	hipMemcpy(hostDisparity, devDisparity,
		hostWidth*hostHeight * sizeof(float),
		hipMemcpyDeviceToHost);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float deltaTime;
	hipEventElapsedTime(&deltaTime, start, stop);
	std::cout << "ʱ����� ��ʱ: " << deltaTime << "ms" << std::endl;
}

void hostOutput()
{
	std::ofstream file;
	file.open("Disparity.ppm", std::ios::out);
	file << "P3" << std::endl;
	file << hostWidth*3 << " " << hostHeight << std::endl;
	file << 255 << std::endl;
	for (int y = 0; y < hostHeight;++y)
	{

		for (int x = 0; x < hostWidth; ++x)
		{
			auto index = y*hostWidth + x;
			auto v = (f4)hostLeft[index];// / (float)hostDint *255.f;
			file << clampIndex((int)v.x, 0, 255) << ' ' <<
				clampIndex((int)v.y, 0, 255) << ' ' <<
				clampIndex((int)v.z, 0, 255) << ' ';

		}

		for (int x = 0; x < hostWidth; ++x)
		{
			auto index = y*hostWidth + x;
			auto v = (f4)hostRight[index];// / (float)hostDint *255.f;
			file << clampIndex((int)v.x, 0, 255) << ' ' <<
				clampIndex((int)v.y, 0, 255) << ' ' <<
				clampIndex((int)v.z, 0, 255) << ' ';

		}

		for (int x = 0; x < hostWidth; ++x)
		{
			auto index = y*hostWidth + x;
			float v = (float)hostDisparity[index];// / (float)hostDint *255.f;
			file << clampIndex((int)v, 0, 255) << ' ' <<
				clampIndex((int)v, 0, 255) << ' ' <<
				clampIndex((int)v, 0, 255) << ' ';

		}


		file << std::endl;

	}
	file.close();
}

int main()
{
	//����ͼƬ
	hostLeft = loadImage("images/left.png", hostWidth, hostHeight);
	hostRight = loadImage("images/right.png", hostWidth, hostHeight);
	hostDint = 100;

//	hipDeviceProp_t prop;
//	hipGetDeviceProperties(&prop, 0);

	///////////////////////////////////////////////
	createGPUBuffer();
	///////////////////////////////////////////////

	hostCalCost();
//	hostCalInterCost();
	hostDP(devImageCost);
	hostCalDisparity();
	hostOutput();
	
	
	////////////////////////////////////////////////
	freeGPUBuffer();
	////////////////////////////////////////////////


	system("pause");
	return 0;
	
}

